#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <math.h>

#include <time.h>
#include <hip/hip_runtime.h>
using namespace std;

#define SIZE_DARR 1000000
float hres[SIZE_DARR] = { 0 };
float hh[1] = { 0 };
float* dres, *dh;
float gpuTime;

int threadsPerBlock = 1024;
int blocksPerGrid = (SIZE_DARR + threadsPerBlock - 1) / threadsPerBlock;


__global__ void CalcIntegralGPU(int n, float* dres, float* dh)
{
	float a = 0.0;
	float b = 1.0;
	float x = 0.0;

	dh[0] = (b - a) / n;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	while (tid < n)
	{
		x = a + dh[0] * (tid + 0.5);
		dres[tid] += (exp(x) + exp(-x)) / 2;
		tid += blockDim.x * gridDim.x;		
	}	

}

float InFunction(float x) //��������������� �������
{
	return (exp(x) + exp(-x)) / 2;
}

float CalcIntegral(int n)
{
	int i;
	float sum, h;
	float a = 0.0;
	float b = 1.0;

	sum = 0.0;

	// n - ���������� �������� ��������������
	h = (b - a) / n; //��� �����

	for (i = 0; i < n; i++) {
		sum += InFunction(a + h * (i + 0.5)); //��������� � ������� ����� � ��������� � �����
	}

	sum *= h;

	return sum;
}

void experiment(int n)
{
	hres[SIZE_DARR] = { 0 };
	hh[1] = { 0 };

	hipMalloc((void**)&dres, sizeof(float) * SIZE_DARR);
	hipMalloc((void**)&dh, sizeof(float));
	hipMemcpy(dres, hres, sizeof(float) * SIZE_DARR, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(dh, hh, sizeof(float) * SIZE_DARR, hipMemcpyKind::hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	gpuTime = 0.0f;

	cout << endl << n << " ���������" << endl << "����� (ms) \n";

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
		
	CalcIntegralGPU<<<blocksPerGrid, threadsPerBlock>>> (n, dres, dh);
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipDeviceSynchronize();

	hipEventElapsedTime(&gpuTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	hipMemcpy(hres, dres, sizeof(float) * SIZE_DARR, hipMemcpyKind::hipMemcpyDeviceToHost);
	hipMemcpy(hh, dh, sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);
	
	float result = 0.0f;
	for (int i = 0; i < n; i++)
	{
		result += hres[i];
	}
	result *= hh[0];

	cout.width(10);
	cout.setf(ios::right);
	cout << gpuTime << endl;

	hipFree(dres);
	hipFree(dh);
}

int main()
{
	setlocale(LC_CTYPE, "rus");
	long long freq = CLOCKS_PER_SEC;

	for (int i = 100; i <= SIZE_DARR; i *= 10)
	{
		long long st = clock();
		CalcIntegral(i);
		st = clock() - st;
		cout << endl << i << " ���������" << endl << "����� (ms) \n";
		cout.width(10);
		cout.setf(ios::right);
		cout << 1000 * st / freq << endl;
	}


	cout << endl << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << endl;

	for (int i = 100; i <= SIZE_DARR; i *= 10)
	{
		experiment(i);
	}

	getchar();
    return 0;
}